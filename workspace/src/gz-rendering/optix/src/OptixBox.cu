#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optixu/optixu_aabb.h>

// box properties
rtDeclareVariable(float3, scale, , );

// ray properties
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, shadingTangent, attribute shadingTangent, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

static __inline__ __device__ void GetVectors(const float3 &_t0,
    const float3 &_t1, float _t, float3 &_norm, float3 &_tang)
{
  // match to min normal
  float3 neg = make_float3(_t == _t0.x ? 1 : 0,
                           _t == _t0.y ? 1 : 0,
                           _t == _t0.z ? 1 : 0);

  // match to max normal
  float3 pos = make_float3(_t == _t1.x ? 1 : 0,
                           _t == _t1.y ? 1 : 0,
                           _t == _t1.z ? 1 : 0);

  // compute final normal
  _norm = pos - neg;

  // compute normal tangent
  if (_norm.x !=  0) _tang.y =  _norm.x;
  if (_norm.y !=  0) _tang.x = -_norm.y;
  if (_norm.z !=  0) _tang.y = -_norm.z;
}

static __inline__ __device__ float2 GetTextureCoordinate(const float3 &_p,
    const float3 &_n)
{
  float u = 0.0;
  float v = 0.0;

  if (_n.x == 1 || _n.x == -1)
  {
    u = _n.x * _p.y / scale.y;
    v = _p.z / scale.z;
  }
  else if (_n.y == 1 || _n.y == -1)
  {
    u = _n.y * _p.x / scale.x;
    v = _p.z / scale.z;
  }
  else if (_n.z == 1 || _n.z == -1)
  {
    u = _n.z * _p.x / scale.x;
    v = _p.y / scale.y;
  }

  return make_float2(u, v) + 0.5;
}

static __inline__ __device__ bool ReportPotentialIntersect(float3 _t0,
    float3 _t1, float _t)
{
  if (rtPotentialIntersection(_t))
  {
    float3 normal, tangent;
    GetVectors(_t0, _t1, _t, normal, tangent);

    shadingNormal = geometricNormal = normal;
    shadingTangent = tangent;

    float3 hitPoint = _t * ray.direction + ray.origin;
    texCoord = GetTextureCoordinate(hitPoint, normal);

    return rtReportIntersection(0);
  }

  return false;
}

RT_PROGRAM void Intersect(int)
{
  // get time to each extrema
  float3 ex = scale / 2;
  float3 t0 = (-ex - ray.origin) / ray.direction;
  float3 t1 = ( ex - ray.origin) / ray.direction;

  // determine extrema times per coord
  float3 near = fminf(t0, t1);
  float3 far  = fmaxf(t0, t1);

  // determine extrema times
  float tmin = fmaxf(near);
  float tmax = fminf(far);

  if (tmin <= tmax)
  {
    if (!ReportPotentialIntersect(t0, t1, tmin))
    {
      ReportPotentialIntersect(t0, t1, tmax);
    }
  }
}

RT_PROGRAM void Bounds(int, float _bounds[6])
{
  float3 ex = scale / 2;
  optix::Aabb* aabb = (optix::Aabb*)_bounds;
  aabb->set(-ex, ex);
}
