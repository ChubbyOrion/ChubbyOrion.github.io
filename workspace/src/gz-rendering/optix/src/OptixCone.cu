#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_aabb.h>

rtDeclareVariable(float3, scale, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, shadingTangent, attribute shadingTangent, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

static __inline__ __device__ bool ReportPotentialIntersect(float _t,
    const float3 &_norm, const float3 &_tang, const float2 &_uv)
{
  if (rtPotentialIntersection(_t))
  {
    shadingNormal = geometricNormal = _norm;
    shadingTangent = _tang;
    texCoord = _uv;
    return rtReportIntersection(0);
  }

  return false;
}

RT_PROGRAM void Intersect(int)
{
  float height = scale.z;
  float radius = scale.x / 2; // TODO: handle scale.y

  float3 direction = ray.direction;
  float3 origin = ray.origin;
  origin.z -= scale.z / 2;

  float k = radius / height;
  float k2 = k * k;

  float a = direction.x * direction.x + direction.y * direction.y - direction.z
    * direction.z * k2;

  float b = 2 * (origin.x * direction.x + origin.y * direction.y - origin.z *
      direction.z * k2);

  float c = origin.x * origin.x + origin.y * origin.y - origin.z * origin.z *
    k2;

  float s = b * b - 4 * a * c;

  if (s < 0)
  {
    return;
  }

  float t1 = (-b + sqrt(s)) / (2 * a);
  float t2 = (-b - sqrt(s)) / (2 * a);

  float3 p1 = origin + t1 * direction;
  float3 p2 = origin + t2 * direction;

  float ymin = fminf(p1.z, p2.z);
  float ymax = fmaxf(p1.z, p2.z);

  if (ymin > 0 || ymax < -height || (ymin < -height && ymax > 0))
  {
    return;
  }

  float3 p = (t1 < t2) ? p1 : p2;
  float3 n;
  float3 tg;
  float2 uv;
  float t;

  float r2 = radius * radius;

  if (p.z > 0)
  {
    p = (t1 < t2) ? p2 : p1;
    t = (t1 < t2) ? t2 : t1;
    float r = radius * p.z / height;
    n = p - make_float3(0, 0, r * k + p.z);
    n = normalize(n);

    float xt = (n.y > 0) ? -n.y :  n.y;
    float yt = (n.x > 0) ?  n.x : -n.x;
    tg = normalize(make_float3(xt, yt, 0));

    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = p.z;
  }
  else if (p.z <= -height)
  {
    t = (-height - origin.z) / direction.z;
    p = origin + t * direction;
    n = make_float3(0, 0, -1);
    tg = make_float3(0, 1, 0);

    if (p.x * p.x + p.y * p.y > r2)
    {
      return;
    }

    uv.x = 0.5 - p.y / scale.y;
    uv.y = 0.5 - p.x / scale.x;
  }
  else
  {
    t = fminf(t1, t2);
    float r = radius * p.z / height;
    n = p - make_float3(0, 0, r * k + p.z);
    n = normalize(n);

    float xt = (n.y > 0) ? -n.y :  n.y;
    float yt = (n.x > 0) ?  n.x : -n.x;
    tg = normalize(make_float3(xt, yt, 0));

    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = p.z;
  }

  uv = uv + 0.5;
  ReportPotentialIntersect(t, n, tg, uv);
}

RT_PROGRAM void Bounds(int, float _result[6])
{
  float3 ex = scale / 2;
  optix::Aabb* aabb = (optix::Aabb*)_result;
  aabb->set(-ex, ex);
}
