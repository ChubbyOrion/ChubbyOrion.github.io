#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_aabb.h>

rtDeclareVariable(float3, scale, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, shadingTangent, attribute shadingTangent, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

static __inline__ __device__ bool ReportPotentialIntersect(float _t)
{
  if (rtPotentialIntersection(_t))
  {
    float radius = scale.x / 2; // TODO: handle scale.y
    float3 normal = (_t * ray.direction + ray.origin) / radius;
    shadingNormal = geometricNormal = normal;

    float xt = (shadingNormal.y > 0) ? -shadingNormal.y :  shadingNormal.y;
    float yt = (shadingNormal.x > 0) ?  shadingNormal.x : -shadingNormal.x;
    shadingTangent = normalize(make_float3(xt, yt, 0));

    float u = atan2(normal.y, normal.x) / M_PI;
    float v = acos(normal.z) / M_PI;
    texCoord = make_float2(u, v) + 0.5;

    return rtReportIntersection(0);
  }

  return false;
}

RT_PROGRAM void Intersect(int)
{
  float3 origin = ray.origin;
  float3 direction = ray.direction;

  float radius = scale.x / 2; // TODO: handle scale.y
  float zzz = dot(origin, direction);
  float aaa = dot(origin, origin) - radius * radius;
  float squaredDistance = zzz * zzz - aaa;

  if (squaredDistance > 0.0f)
  {
    float distance = sqrtf(squaredDistance);
    float t = -zzz - distance;

    if (!ReportPotentialIntersect(t))
    {
      float t = -zzz + distance;
      ReportPotentialIntersect(t);
    }
  }
}

RT_PROGRAM void Bounds(int, float _result[6])
{
  float3 ex = scale / 2;
  optix::Aabb* aabb = (optix::Aabb*)_result;
  aabb->set(-ex, ex);
}
