#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_aabb.h>

rtBuffer<float3> vertexBuffer;
rtBuffer<float3> normalBuffer;
rtBuffer<float2> texCoordBuffer;
rtBuffer<int3>   indexBuffer;
rtDeclareVariable(float3, scale, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, shadingTangent, attribute shadingTangent, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

RT_PROGRAM void Intersect(int primIndex)
{
  int3 indices = indexBuffer[primIndex];

  float3 p0 = scale * vertexBuffer[ indices.x ];
  float3 p1 = scale * vertexBuffer[ indices.y ];
  float3 p2 = scale * vertexBuffer[ indices.z ];

  float3 n;
  float t, beta, gamma;

  if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma))
  {

    if (rtPotentialIntersection(t))
    {
      if (normalBuffer.size() == 0 || indices.x < 0 || indices.y < 0 ||
          indices.z < 0 )
      {
        shadingNormal = normalize(n);
      }
      else
      {
        float3 n0 = normalBuffer[ indices.x ] / scale;
        float3 n1 = normalBuffer[ indices.y ] / scale;
        float3 n2 = normalBuffer[ indices.z ] / scale;

        shadingNormal = normalize(n1 * beta + n2 * gamma + n0 *
          (1.0f - beta - gamma));
      }

      geometricNormal = normalize(n);

      if ( texCoordBuffer.size() == 0 || indices.x < 0 || indices.y < 0 ||
        indices.z < 0 )
      {
        texCoord = make_float2( 0.0f, 0.0f );
        shadingTangent = make_float3(0);
      }
      else
      {
        float2 t0 = texCoordBuffer[ indices.x ];
        float2 t1 = texCoordBuffer[ indices.y ];
        float2 t2 = texCoordBuffer[ indices.z ];

        texCoord = t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma);
        shadingTangent = make_float3(0); // TODO: implement
      }

      rtReportIntersection(0);
    }

  }
}

RT_PROGRAM void Bounds(int _primIndex, float _result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)_result;
  const int3 indices = indexBuffer[_primIndex];

  const float3 v0   = vertexBuffer[ indices.x ];
  const float3 v1   = vertexBuffer[ indices.y ];
  const float3 v2   = vertexBuffer[ indices.z ];
  const float  area = length(cross(v1 - v0, v2 - v0));

  if(area > 0.0f && !isinf(area))
  {
    aabb->m_min = scale * fminf(fminf(v0, v1), v2 );
    aabb->m_max = scale * fmaxf(fmaxf(v0, v1), v2 );
  }
  else
  {
    aabb->invalidate();
  }
}
